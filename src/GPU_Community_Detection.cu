#include "hip/hip_runtime.h"
#include <GPU_Community_Detection.cuh>
using namespace std;

static int CD_GRAPHSIZE;
static vector<int> row_ptr, col_indices;
static vector<int> neighbor;
static int* row_ptr_gpu;
static int* labels_gpu, * neighbor_gpu;
static int* reduce_label, * reduce_label_count;
static int* updating;

template <typename T>
void make_csr(graph_structure<T> &graph, int& CD_GRAPHSIZE)
{
    CD_GRAPHSIZE = graph.size();
    row_ptr.resize(CD_GRAPHSIZE + 1);
    row_ptr[0] = 0;
    CSR_graph<T> ARRAY_graph;
    ARRAY_graph=graph.toCSR();
    row_ptr=ARRAY_graph.OUTs_Neighbor_start_pointers;
    neighbor=ARRAY_graph.OUTs_Edges;
    col_indices=neighbor;
    // for (int i = 0; i < CD_GRAPHSIZE; i++)
    // {
    //     for (auto& edge : graph.OUTs[i])
    //     {
    //         int neighbor_vertex = edge.first;
    //         neighbor.push_back(neighbor_vertex);
    //         col_indices.push_back(neighbor_vertex);
    //     }
    //     row_ptr[i + 1] = row_ptr[i] + graph.ADJs[i].size();
    // }
}


__global__ void init_label(int* labels_gpu,int CD_GRAPHSIZE)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= 0 && tid < CD_GRAPHSIZE)
    {
        labels_gpu[tid] = tid;
    }
}

__global__ void LPA(int* row_ptr_gpu, int* labels_gpu, int* neighbor_gpu, int* reduce_label, int* reduce_label_count,int CD_GRAPHSIZE,int BLOCK_PER_VER)
{
    extern __shared__ int label_counts[];
    extern __shared__ int label[];
    int ver = blockIdx.x / BLOCK_PER_VER;
    int tid = (blockIdx.x % BLOCK_PER_VER) * blockDim.x + threadIdx.x;
    int segment_order = blockIdx.x % BLOCK_PER_VER;
    int stid = threadIdx.x;
    if (stid == ver)
    {
        label_counts[stid] = 1;
    }
    else
    {
        label_counts[stid] = 0;
    }
    label[stid] = tid;

    __syncthreads();

    int start = row_ptr_gpu[ver], end = row_ptr_gpu[ver + 1];
    if (tid >= end - start)
    {
        return;
    }
    int neighbor_label = labels_gpu[neighbor_gpu[start + tid]];
    if (neighbor_label >= segment_order * CD_THREAD_PER_BLOCK && neighbor_label < (segment_order + 1) * CD_THREAD_PER_BLOCK)
        atomicAdd(&label_counts[neighbor_label - segment_order * CD_THREAD_PER_BLOCK], 1);

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            if (label_counts[tid] < label_counts[tid + s])
            {
                label_counts[tid] = label_counts[tid + s];
                label[tid] = label_counts[tid + s];
            }
            else if (label_counts[tid] == label_counts[tid + s] && label[tid] > label_counts[tid + s])
            {
                label[tid] = label_counts[tid + s];
            }
        }
        __syncthreads();
    }
    reduce_label_count[blockIdx.x] = label_counts[0];
    reduce_label[blockIdx.x] = label[0];
    return;
}

__global__ void Updating_label(int* reduce_label, int* reduce_label_count, int* updating, int* labels_gpu,int CD_GRAPHSIZE,int BLOCK_PER_VER)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= CD_GRAPHSIZE)
        return;
    int cont = 1, label = labels_gpu[tid];
    int start = tid * BLOCK_PER_VER, end = start + BLOCK_PER_VER;
    for (int i = start; i < end; ++i)
    {
        if (reduce_label_count[i] > cont)
        {
            cont = reduce_label_count[i];
            label = reduce_label[i];
        }
        else if (reduce_label_count[i] == cont && reduce_label[i] < label)
        {
            label = reduce_label[i];
        }
    }
    if (label != labels_gpu[tid])
        *updating = 1;
    labels_gpu[tid] = label;
    return;
}

int Community_Detection(graph_structure<double>& graph, float* elapsedTime)
{
    make_csr(graph,CD_GRAPHSIZE);

    int BLOCK_PER_VER=((CD_GRAPHSIZE + CD_THREAD_PER_BLOCK - 1) / CD_THREAD_PER_BLOCK);
    int REDUCE_BLOCK_PER_GRID=(CD_GRAPHSIZE * BLOCK_PER_VER + CD_THREAD_PER_BLOCK - 1) / CD_THREAD_PER_BLOCK;

    dim3 blockPerGrid((CD_GRAPHSIZE + CD_THREAD_PER_BLOCK - 1) / CD_THREAD_PER_BLOCK, 1, 1);
    dim3 useBlock((CD_GRAPHSIZE + CD_THREAD_PER_BLOCK - 1) / CD_THREAD_PER_BLOCK * CD_GRAPHSIZE, 1, 1);
    dim3 threadPerBlock(CD_THREAD_PER_BLOCK, 1, 1);
    dim3 reduceBlock(REDUCE_BLOCK_PER_GRID, 1, 1);

    hipMalloc(&row_ptr_gpu, (CD_GRAPHSIZE + 1) * sizeof(int));
    hipMalloc(&labels_gpu, CD_GRAPHSIZE * sizeof(int));
    hipMalloc(&neighbor_gpu, neighbor.size() * sizeof(int));
    hipMalloc(&reduce_label, CD_GRAPHSIZE * BLOCK_PER_VER * sizeof(int));
    hipMalloc(&reduce_label_count, CD_GRAPHSIZE * BLOCK_PER_VER * sizeof(int));
    hipMemcpy(row_ptr_gpu, row_ptr.data(), row_ptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(neighbor_gpu, neighbor.data(), neighbor.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMallocManaged(&updating, sizeof(int));
    
    int it=0;
    *updating = 1;
    init_label << <blockPerGrid, threadPerBlock >> > (labels_gpu,CD_GRAPHSIZE);
    hipDeviceSynchronize();
    hipEvent_t GPUstart, GPUstop;
    hipEventCreate(&GPUstart);
    hipEventCreate(&GPUstop);
    hipEventRecord(GPUstart, 0);
    while (*updating)
    {
        it++;
        *updating = 0;
        LPA << <useBlock, threadPerBlock, sizeof(int)* CD_THREAD_PER_BLOCK >> > (row_ptr_gpu, labels_gpu, neighbor_gpu, reduce_label, reduce_label_count,CD_GRAPHSIZE,BLOCK_PER_VER);
        hipDeviceSynchronize();
        Updating_label << <reduceBlock, threadPerBlock >> > (reduce_label, reduce_label_count, updating, labels_gpu,CD_GRAPHSIZE,BLOCK_PER_VER);
        hipDeviceSynchronize();
    }

    hipEventRecord(GPUstop, 0);
    hipEventSynchronize(GPUstop);


    hipEventElapsedTime(elapsedTime, GPUstart, GPUstop);

    hipEventDestroy(GPUstart);
    hipEventDestroy(GPUstop);

    hipFree(row_ptr_gpu);
    hipFree(labels_gpu);
    hipFree(neighbor_gpu);
    hipFree(reduce_label);
    hipFree(reduce_label_count);

    return 0;
}
